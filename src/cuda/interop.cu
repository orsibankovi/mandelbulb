#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "interop.cuh"
#include "stdio.h"
#include <memory>
#include <cuda/std/complex>

#define checkCudaError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    }
}

uint32_t imageWidth, imageHeight;
float _zoom = 1.0f;
hipExternalMemory_t cudaExtMemImageBuffer; // memory handler to the imported memory allocation
hipMipmappedArray_t cudaMipmappedImageArray; // the image interpreted as a mipmapped array
hipSurfaceObject_t surfaceObject; // surface object to the first mip level of the array. Allows write

void freeExportedVulkanImage()
{
    checkCudaError(hipDestroySurfaceObject(surfaceObject));
    checkCudaError(hipFreeMipmappedArray(cudaMipmappedImageArray));
    checkCudaError(hipDestroyExternalMemory(cudaExtMemImageBuffer));
}

void exportVulkanImageToCuda_R8G8B8A8Unorm(void* mem, VkDeviceSize size, VkDeviceSize offset, uint32_t width, uint32_t height)
{
    imageWidth = width;
    imageHeight = height;

    // import memory into cuda through native handle (win32)
    hipExternalMemoryHandleDesc cudaExtMemHandleDesc;
    memset(&cudaExtMemHandleDesc, 0, sizeof(cudaExtMemHandleDesc));
    cudaExtMemHandleDesc.type = hipExternalMemoryHandleTypeOpaqueWin32; // after win8
    cudaExtMemHandleDesc.handle.win32.handle = mem; // allocation handle
    cudaExtMemHandleDesc.size = size; // allocation size
   
    checkCudaError(hipImportExternalMemory(&cudaExtMemImageBuffer, &cudaExtMemHandleDesc));

    // extract mipmapped array from memory
    cudaExternalMemoryMipmappedArrayDesc externalMemoryMipmappedArrayDesc;
    memset(&externalMemoryMipmappedArrayDesc, 0, sizeof(externalMemoryMipmappedArrayDesc));

    // we want ot interpret the raw memory as an image so we need to specify its format and layout
    hipExtent extent = make_hipExtent(width, height, 0);
    hipChannelFormatDesc formatDesc; // 4 channel, 8 bit per channel, unsigned
    formatDesc.x = 8;
    formatDesc.y = 8;
    formatDesc.z = 8;
    formatDesc.w = 8;
    formatDesc.f = hipChannelFormatKindUnsigned;

    externalMemoryMipmappedArrayDesc.offset = offset; // the image starts here
    externalMemoryMipmappedArrayDesc.formatDesc = formatDesc;
    externalMemoryMipmappedArrayDesc.extent = extent;
    externalMemoryMipmappedArrayDesc.flags = 0;
    externalMemoryMipmappedArrayDesc.numLevels = 1; // no mipmapping
    checkCudaError(cudaExternalMemoryGetMappedMipmappedArray(&cudaMipmappedImageArray, cudaExtMemImageBuffer, &externalMemoryMipmappedArrayDesc));

    // extract first level
    hipArray_t cudaMipLevelArray;
    checkCudaError(hipGetMipmappedArrayLevel(&cudaMipLevelArray, cudaMipmappedImageArray, 0));

    // create surface object for writing
    hipResourceDesc resourceDesc;
    memset(&resourceDesc, 0, sizeof(resourceDesc));
    resourceDesc.resType = hipResourceTypeArray;
    resourceDesc.res.array.array = cudaMipLevelArray;
    
    checkCudaError(hipCreateSurfaceObject(&surfaceObject, &resourceDesc));
}

hipExternalSemaphore_t cudaWaitsForVulkanSemaphore, vulkanWaitsForCudaSemaphore;

void freeExportedSemaphores()
{
    checkCudaError(hipDestroyExternalSemaphore(cudaWaitsForVulkanSemaphore));
    checkCudaError(hipDestroyExternalSemaphore(vulkanWaitsForCudaSemaphore));
}

void exportSemaphoresToCuda(void* cudaWaitsForVulkanSemaphoreHandle, void* vulkanWaitsForCudaSemaphoreHandle) {
    hipExternalSemaphoreHandleDesc externalSemaphoreHandleDesc;
    memset(&externalSemaphoreHandleDesc, 0, sizeof(externalSemaphoreHandleDesc));
    externalSemaphoreHandleDesc.flags = 0;
    externalSemaphoreHandleDesc.type = hipExternalSemaphoreHandleTypeOpaqueWin32;

    externalSemaphoreHandleDesc.handle.win32.handle = cudaWaitsForVulkanSemaphoreHandle;
    checkCudaError(hipImportExternalSemaphore(&cudaWaitsForVulkanSemaphore, &externalSemaphoreHandleDesc));

    externalSemaphoreHandleDesc.handle.win32.handle = vulkanWaitsForCudaSemaphoreHandle;
    checkCudaError(hipImportExternalSemaphore(&vulkanWaitsForCudaSemaphore, &externalSemaphoreHandleDesc));
}

// compresses 4 32bit floats into a 32bit uint
__device__ unsigned int rgbaFloatToInt(float4 rgba) {
    rgba.x = __saturatef(rgba.x);  // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return ((unsigned int)(rgba.w * 255.0f) << 24) |
        ((unsigned int)(rgba.z * 255.0f) << 16) |
        ((unsigned int)(rgba.y * 255.0f) << 8) |
        ((unsigned int)(rgba.x * 255.0f));
}

__device__ float3 operator+(const float3& a, const float3& b) {
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ float3 operator*(const float3& a, const float& b) {
    return make_float3(a.x * b, a.y * b, a.z * b);
}

__device__ float length(const float3& vec) {
    return sqrt(vec.x * vec.x + vec.y * vec.y + vec.z * vec.z);
}

__device__ float3 normalize(const float3 vec) {
    float inverted_len = 1.0f / length(vec);
    return vec * inverted_len;
}

typedef struct {
    float3 o;
    float3 d;
} ray;

__device__ ray get_ray(const float& u, const float& v, float zoom, float offsetX, float offsetY) {
    ray r;
    r.o = make_float3(-3.0, 0.1, 0.1);
    r.d = normalize(make_float3(1.0 * zoom, u + offsetX / 1000, v + offsetY / 1000));
    return r;
}

__device__ float mandelbulb(float3 pos, float3 rotation) {
    float3 z = pos;

    // Rotate around X axis
    float temp_y = z.y;
    z.y = z.y * cos(rotation.x) - z.z * sin(rotation.x);
    z.z = temp_y * sin(rotation.x) + z.z * cos(rotation.x);

    // Rotate around Y axis
    float temp_x = z.x;
    z.x = z.x * cos(rotation.y) + z.z * sin(rotation.y);
    z.z = -temp_x * sin(rotation.y) + z.z * cos(rotation.y);

    // Rotate around Z axis
    temp_x = z.x;
    z.x = z.x * cos(rotation.z) - z.y * sin(rotation.z);
    z.y = temp_x * sin(rotation.z) + z.y * cos(rotation.z);

    float dr = 1.0;
    float r = 0.0;
    int iterations = 16;
    float Bailout = 4.0;
    float n = 12.0;

    for (int i = 0; i < iterations; i++) {
        r = length(z);
        if (r > Bailout) break;

        // convert to spherical coordinates
        float theta = acos(z.z / r);
        float phi = atan2(z.y, z.x);
        dr = powf(r, n - 1.0) * n * dr + 1.0;

        // scale and rotate the point
        float zr = pow(r, n);
        theta = theta * n;
        phi = phi * n;

        // convert back to cartesian coordinates
        z = make_float3(sin(theta) * cos(phi), sin(phi) * sin(theta), cos(theta)) * zr;

        z = z + pos;
    }
    return 0.5 * log(r) * r / dr;
}

__device__ float march(ray r, float3 rotation) {
    float total_dist = 0.0;
    int max_ray_steps = 48;
    float min_distance = 0.0005;

    int steps;
    for (steps = 0; steps < max_ray_steps; ++steps) {
        float3 p = r.o + r.d * total_dist;
        float distance = mandelbulb(p, rotation);
        total_dist += distance;
        if (distance < min_distance) break;
    }
    return 1.0 - (float)steps / (float)max_ray_steps;
}


__global__ void MandelbulbDraw(hipSurfaceObject_t dstSurface, size_t width, size_t height, float zoom, float offsetX, float offsetY, float3 rotation)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

    float min_w_h = (float) min(width, height);

    float ar = (float)width / (float)height;
    float u = (float)x / min_w_h - ar * 0.5f;
    float v = (float)y / min_w_h - 0.5f;

    ray r = get_ray(u, v, zoom, offsetX, offsetY);
    float c = march(r, rotation);

	float4 dataOut = make_float4(c * 0.5f, c * 0.5f, c, 1.0f);

	surf2Dwrite(rgbaFloatToInt(dataOut), dstSurface, x * 4, y);
}

void renderCuda(float zoom, float offsetX, float offsetY, float t1, float t2, float t3)
{
    uint32_t nthreads = 32;
    dim3 dimGrid{ imageWidth / nthreads + 1, imageHeight / nthreads + 1};
    dim3 dimBlock{ nthreads, nthreads };
    float3 rotation = make_float3(t1, t2, t3);
    MandelbulbDraw << <dimGrid, dimBlock >> > (surfaceObject, imageWidth, imageHeight, zoom, offsetX, offsetY, rotation);
    checkCudaError(hipGetLastError());
    checkCudaError(hipDeviceSynchronize()); // not optimal! should be synced with vulkan using semaphores
}