#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "interop.cuh"
#include "stdio.h"
#include <memory>
#include <cuda/std/complex>

#define checkCudaError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    }
}

uint32_t imageWidth, imageHeight;
float _zoom = 1.0f;
hipExternalMemory_t cudaExtMemImageBuffer; // memory handler to the imported memory allocation
hipMipmappedArray_t cudaMipmappedImageArray; // the image interpreted as a mipmapped array
hipSurfaceObject_t surfaceObject; // surface object to the first mip level of the array. Allows write

void freeExportedVulkanImage()
{
    checkCudaError(hipDestroySurfaceObject(surfaceObject));
    checkCudaError(hipFreeMipmappedArray(cudaMipmappedImageArray));
    checkCudaError(hipDestroyExternalMemory(cudaExtMemImageBuffer));
}

void exportVulkanImageToCuda_R8G8B8A8Unorm(void* mem, VkDeviceSize size, VkDeviceSize offset, uint32_t width, uint32_t height)
{
    imageWidth = width;
    imageHeight = height;

    // import memory into cuda through native handle (win32)
    hipExternalMemoryHandleDesc cudaExtMemHandleDesc;
    memset(&cudaExtMemHandleDesc, 0, sizeof(cudaExtMemHandleDesc));
    cudaExtMemHandleDesc.type = hipExternalMemoryHandleTypeOpaqueWin32; // after win8
    cudaExtMemHandleDesc.handle.win32.handle = mem; // allocation handle
    cudaExtMemHandleDesc.size = size; // allocation size
   
    checkCudaError(hipImportExternalMemory(&cudaExtMemImageBuffer, &cudaExtMemHandleDesc));

    // extract mipmapped array from memory
    cudaExternalMemoryMipmappedArrayDesc externalMemoryMipmappedArrayDesc;
    memset(&externalMemoryMipmappedArrayDesc, 0, sizeof(externalMemoryMipmappedArrayDesc));

    // we want ot interpret the raw memory as an image so we need to specify its format and layout
    hipExtent extent = make_hipExtent(width, height, 0);
    hipChannelFormatDesc formatDesc; // 4 channel, 8 bit per channel, unsigned
    formatDesc.x = 8;
    formatDesc.y = 8;
    formatDesc.z = 8;
    formatDesc.w = 8;
    formatDesc.f = hipChannelFormatKindUnsigned;

    externalMemoryMipmappedArrayDesc.offset = offset; // the image starts here
    externalMemoryMipmappedArrayDesc.formatDesc = formatDesc;
    externalMemoryMipmappedArrayDesc.extent = extent;
    externalMemoryMipmappedArrayDesc.flags = 0;
    externalMemoryMipmappedArrayDesc.numLevels = 1; // no mipmapping
    checkCudaError(cudaExternalMemoryGetMappedMipmappedArray(&cudaMipmappedImageArray, cudaExtMemImageBuffer, &externalMemoryMipmappedArrayDesc));

    // extract first level
    hipArray_t cudaMipLevelArray;
    checkCudaError(hipGetMipmappedArrayLevel(&cudaMipLevelArray, cudaMipmappedImageArray, 0));

    // create surface object for writing
    hipResourceDesc resourceDesc;
    memset(&resourceDesc, 0, sizeof(resourceDesc));
    resourceDesc.resType = hipResourceTypeArray;
    resourceDesc.res.array.array = cudaMipLevelArray;
    
    checkCudaError(hipCreateSurfaceObject(&surfaceObject, &resourceDesc));
}

hipExternalSemaphore_t cudaWaitsForVulkanSemaphore, vulkanWaitsForCudaSemaphore;

void freeExportedSemaphores()
{
    checkCudaError(hipDestroyExternalSemaphore(cudaWaitsForVulkanSemaphore));
    checkCudaError(hipDestroyExternalSemaphore(vulkanWaitsForCudaSemaphore));
}

void exportSemaphoresToCuda(void* cudaWaitsForVulkanSemaphoreHandle, void* vulkanWaitsForCudaSemaphoreHandle) {
    hipExternalSemaphoreHandleDesc externalSemaphoreHandleDesc;
    memset(&externalSemaphoreHandleDesc, 0, sizeof(externalSemaphoreHandleDesc));
    externalSemaphoreHandleDesc.flags = 0;
    externalSemaphoreHandleDesc.type = hipExternalSemaphoreHandleTypeOpaqueWin32;

    externalSemaphoreHandleDesc.handle.win32.handle = cudaWaitsForVulkanSemaphoreHandle;
    checkCudaError(hipImportExternalSemaphore(&cudaWaitsForVulkanSemaphore, &externalSemaphoreHandleDesc));

    externalSemaphoreHandleDesc.handle.win32.handle = vulkanWaitsForCudaSemaphoreHandle;
    checkCudaError(hipImportExternalSemaphore(&vulkanWaitsForCudaSemaphore, &externalSemaphoreHandleDesc));
}

// compresses 4 32bit floats into a 32bit uint
__device__ unsigned int rgbaFloatToInt(float4 rgba) {
    rgba.x = __saturatef(rgba.x);  // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return ((unsigned int)(rgba.w * 255.0f) << 24) |
        ((unsigned int)(rgba.z * 255.0f) << 16) |
        ((unsigned int)(rgba.y * 255.0f) << 8) |
        ((unsigned int)(rgba.x * 255.0f));
}

__device__ float3 operator+(const float3& a, const float3& b) {
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ float3 operator*(const float3& a, const float& b) {
    return make_float3(a.x * b, a.y * b, a.z * b);
}

__device__ float length(const float3& vec) {
    return sqrt(vec.x * vec.x + vec.y * vec.y + vec.z * vec.z);
}

__device__ float3 normalize(const float3 vec) {
    float inverted_len = 1.0f / length(vec);
    return vec * inverted_len;
}

typedef struct {
    float3 origin;
    float3 direction;
} ray;

__device__ ray get_ray(const float& u, const float& v, float zoom, float offsetX, float offsetY) {
    ray r;
    r.origin = make_float3(-3.5, 0.0, 0.0);
    r.direction = normalize(make_float3(1.0 * zoom, u + offsetX, v + offsetY));
    return r;
}

__device__ float mandelbulb(float3 pos, float3 rotation) {
    float3 z = pos;

    // Rotate around X axis
    float temp_y = z.y;
    z.y = z.y * cos(rotation.x) - z.z * sin(rotation.x);
    z.z = temp_y * sin(rotation.x) + z.z * cos(rotation.x);

    // Rotate around Y axis
    float temp_x = z.x;
    z.x = z.x * cos(rotation.y) - z.z * sin(rotation.y);
    z.z = temp_x * sin(rotation.y) + z.z * cos(rotation.y);

    // Rotate around Z axis
    temp_x = z.x;
    z.x = z.x * cos(rotation.z) - z.y * sin(rotation.z);
    z.y = temp_x * sin(rotation.z) + z.y * cos(rotation.z);

    float derivative = 1.0;
    float radius = 0.0;
    int iterations = 8;
    float power = 12.0;

    for (int i = 0; i < iterations; i++) {

        radius = length(z);

        if (radius > 2.0) break;

        // convert to spherical coordinates
        float theta = acos(z.z / radius);
        float phi = atan2(z.y, z.x);
        derivative = powf(radius, power - 1.0) * power * derivative + 1.0;

        // scale and rotate the point
        float zr = pow(radius, power);
        theta = theta * power;
        phi = phi * power;

        // convert back to cartesian coordinates
        z = make_float3(sin(theta) * cos(phi), sin(phi) * sin(theta), cos(theta)) * zr;

        z = z + pos;
    }
    return 0.5 * log(radius) * radius / derivative;
}

__device__ float3 calculateNormal(float3 pos, float3 rotation) {
    float epsilon = 0.0005;
    float dx = mandelbulb(make_float3(pos.x + epsilon, pos.y, pos.z), rotation) - mandelbulb(make_float3(pos.x - epsilon, pos.y, pos.z), rotation);
    float dy = mandelbulb(make_float3(pos.x, pos.y + epsilon, pos.z), rotation) - mandelbulb(make_float3(pos.x, pos.y - epsilon, pos.z), rotation);
    float dz = mandelbulb(make_float3(pos.x, pos.y, pos.z + epsilon), rotation) - mandelbulb(make_float3(pos.x, pos.y, pos.z - epsilon), rotation);
    return normalize(make_float3(dx, dy, dz));
}

__device__ float march(ray r, float3 rotation, float3* hitPos) {
    float total_dist = 0.0;
    int max_ray_steps = 48;
    float min_distance = 0.0005;

    int steps;
    for (steps = 0; steps < max_ray_steps; ++steps) {
        float3 p = r.origin + r.direction * total_dist;
        float distance = mandelbulb(p, rotation);
        total_dist += distance;
        if (distance < min_distance) {
            *hitPos = p;
            break;
        }
    }
    return 1.0 - (float)steps / (float)max_ray_steps;
}


__global__ void MandelbulbDraw(hipSurfaceObject_t dstSurface, size_t width, size_t height, float zoom, float offsetX, float offsetY, float3 rotation, bool normal_surface)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    float min_w_h = (float)min(width, height);

    float ar = (float)width / (float)height;
    float u = (float)x / min_w_h - ar * 0.5f;
    float v = (float)y / min_w_h - 0.5f;

    ray r = get_ray(u, v, zoom, offsetX, offsetY);
    float3 hitPos;
    float c = march(r, rotation, &hitPos);

    float4 dataOut = make_float4(c * 0.3f, c * 0.3f, c * 1.5f, 1.0f);

    if (normal_surface) {
        float3 normal = calculateNormal(hitPos, rotation) * 0.6f + make_float3(0.4f, 0.4f, 0.4f);
        dataOut = make_float4(normal.x, normal.y, normal.z, 1.0f);
	}

    surf2Dwrite(rgbaFloatToInt(dataOut), dstSurface, x * 4, y);
}

void renderCuda(float zoom, float offsetX, float offsetY, float t1, float t2, float t3, bool normal_surface)
{
    hipExternalSemaphoreWaitParams extSemaphoreWaitParams;
    memset(&extSemaphoreWaitParams, 0, sizeof(extSemaphoreWaitParams));
    extSemaphoreWaitParams.params.fence.value = 0;
    extSemaphoreWaitParams.flags = 0; checkCudaError(hipWaitExternalSemaphoresAsync(&cudaWaitsForVulkanSemaphore, &extSemaphoreWaitParams, 1));

    uint32_t nthreads = 32;
    dim3 dimGrid{ imageWidth / nthreads + 1, imageHeight / nthreads + 1};
    dim3 dimBlock{ nthreads, nthreads };
    float3 rotation = make_float3(t1, t2, t3);
    MandelbulbDraw << <dimGrid, dimBlock >> > (surfaceObject, imageWidth, imageHeight, zoom, offsetX, offsetY, rotation, normal_surface);
    checkCudaError(hipGetLastError());
    //checkCudaError(hipDeviceSynchronize()); // not optimal! should be synced with vulkan using semaphores

    hipExternalSemaphoreSignalParams extSemaphoreSignalParams;
    memset(&extSemaphoreSignalParams, 0, sizeof(extSemaphoreSignalParams));
    extSemaphoreSignalParams.params.fence.value = 0;
    extSemaphoreSignalParams.flags = 0; checkCudaError(hipSignalExternalSemaphoresAsync(&vulkanWaitsForCudaSemaphore, &extSemaphoreSignalParams, 1));
}